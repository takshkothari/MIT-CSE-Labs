#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__
void word_count_kernel(char* str, char* key, int* word_indices, int* result)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // get idx'th word
    int si = word_indices[idx];
    int ei = word_indices[idx+1];
    char word[100];
     int i = 0;
    for (i = 0; i < (ei-si-1); i++)
        word[i] = str[si+1+i];
    word[i] = '\0';

    // compare word and key
    int i1 = 0;
    int i2 = 0;
    int is_equal = 1;

    while (word[i1] != '\0' && key[i2] != '\0')
    {
        if (word[i1] == key[i2])
        {
            i1++;
            i2++;
        }
        else
        {
            is_equal = 0;
            break;
        }
    }
    if (is_equal == 1)
    {
        atomicAdd(result, 1);
    }
}

int main()
{
    // char str[100] = "We are having PCAP Lab today morning";
    // char key[100] = "today";
    
    char str[100], key[20];
    printf("Enter string: ");
    scanf("%[^\n]s", str);
    printf("Enter key: ");
    scanf("%s", key);

    int str_len = strlen(str), key_len = strlen(key);
    int word_count = 0;

    for (int i = 0; i < str_len; i++)
    {
        if (str[i] == ' ')
            word_count++;
    }
    word_count--;
    int* word_indices = (int*) (malloc(word_count * sizeof(int)));
    int wi = -1;

    for (int i = 0; i < str_len; i++)
    {
        if (str[i] == ' ')
            word_indices[++wi] = i;
    }

    int result = 0;

    char* d_str;
    char* d_key;
    int* d_word_indices;
    int* d_result;

    hipMalloc((void**)&d_str, str_len * sizeof(char));
    hipMalloc((void**)&d_key, key_len * sizeof(char));
    hipMalloc((void**)&d_word_indices, (word_count+1) * sizeof(int));
    hipMalloc((void**)&d_result, sizeof(int));
    
    hipMemcpy(d_str, str, str_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, key_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word_indices, word_indices, (word_count+1) * sizeof(int),     hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);

    word_count_kernel<<<1, word_count>>>(d_str, d_key, d_word_indices, d_result);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Input String: %s\n", str);
    printf("Key: %s\n", key);
    printf("Total occurances of %s is %d\n", key, result);

    hipFree(d_str);
    hipFree(d_key);
    hipFree(d_result);

    return 0;
}