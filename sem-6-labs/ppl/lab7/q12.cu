#include<stdio.h>
#include "hip/hip_runtime.h"


__global__ void vectorAdd_q1A(int *A, int *B,int *C) {
	int tid=blockIdx.x;
		C[tid]=A[tid]+B[tid];
}

__global__ void vectorAdd_q1B(int *A, int *B,int *C) {
	int tid=threadIdx.x;
		C[tid]=A[tid]+B[tid];
}

__global__ void vectorAdd_q2(int *A, int *B,int *C,int n) {
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	if(tid<n)
		C[tid]=A[tid]+B[tid];
}

int main(void) {
	int n;

	printf("Enter number of elements: ");
	scanf("%d",&n);

	int *h_A,*h_B,*h_C;
	int *d_A, *d_B, *d_C;
	
	int size = sizeof(int)*n;

	h_A=(int*)malloc(sizeof(size));
	h_B=(int*)malloc(sizeof(size));
	h_C=(int*)malloc(sizeof(size));

	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	printf("vector 1: ");
	for(int i=0;i<n;i++)
		scanf("%d",&h_A[i]);

	printf("vector 2: ");
	for(int i=0;i<n;i++)
		scanf("%d",&h_B[i]);


	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	
	dim3 dimGrid_q1A(n,1,1);
	dim3 dimBlock_q1A(1,1,1);

	vectorAdd_q1A<<<dimGrid_q1A,dimBlock_q1A>>>(d_A,d_B,d_C);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("result q1a: ");
	for(int i=0;i<n;i++)
		printf("%d ",h_C[i]);

	printf("\n");

	dim3 dimGrid_q1B(1,1,1);
	dim3 dimBlock_q1B(n,1,1);

	vectorAdd_q1B<<<dimGrid_q1B,dimBlock_q1B>>>(d_A,d_B,d_C);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("result q1b: ");
	for(int i=0;i<n;i++)
		printf("%d ",h_C[i]);

	printf("\n");

	dim3 dimGrid_q2(ceil(n/256.0),1,1);
	dim3 dimBlock_q2(256,1,1);

	vectorAdd_q2<<<dimGrid_q2,dimBlock_q2>>>(d_A,d_B,d_C,n);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("result q2: ");
	for(int i=0;i<n;i++)
		printf("%d ",h_C[i]);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}