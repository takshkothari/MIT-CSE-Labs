#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void transpose(int *a, int *t) {
    int n = threadIdx.x, m = blockIdx.x, size = blockDim.x, size1 = gridDim.x, size2 = gridDim.x;
    t[n * size1 + m] = a[m * size + n];
}

int main(void) {
    int *a, *t, m, n, i, j;
    int *d_a, *d_t;
    printf("Enter value of m: ");
    scanf("%d", &m);
    printf("Enter value of n: ");
    scanf("%d", &n);

    int size = sizeof(int) * m * n;

    a = (int *)malloc(size);
    t = (int *)malloc(size);

    printf("Enter input matrix: \n");
    for (i = 0; i < m * n; i++) 
        scanf("%d", &a[i]);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_t, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    transpose<<<m, n>>>(d_a, d_t);

    hipMemcpy(t, d_t, size, hipMemcpyDeviceToHost);
    printf("Result is:\n");
    for (i = 0; i < n; i++) {
        for(j = 0; j < m; j++)
            printf("%d ", t[i * m + j]);
        printf("\n");
    }
    
    getchar();
    hipFree(d_a);
    hipFree(d_t);
    return 0;
}