
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void spmv_kernel(int* row_ptr, int* col_ind, int* values, int* x, int* y, int num_rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_rows) {
        int sum = 0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];

        for (int j = row_start; j < row_end; j++) {
            sum += values[j] * x[col_ind[j]];
        }

        y[row] = sum;
    }
}

int main() {
    int num_rows = 3;
    int num_cols = 3;
    int num_nonzeros = 6;

    int row_ptr[] = {0, 2, 5, 6};
    int col_ind[] = {0, 2, 0, 1, 2, 1};
    int values[] = {1, 2, 3, 4, 5, 6};

    int x[] = {1, 1, 1};
    int y[num_rows];

    int* d_row_ptr, * d_col_ind;
    int* d_values, * d_x, * d_y;

    hipMalloc((void**)&d_row_ptr, (num_rows + 1) * sizeof(int));
    hipMalloc((void**)&d_col_ind, num_nonzeros * sizeof(int));
    hipMalloc((void**)&d_values, num_nonzeros * sizeof(int));
    hipMalloc((void**)&d_x, num_cols * sizeof(int));
    hipMalloc((void**)&d_y, num_rows * sizeof(int));

    hipMemcpy(d_row_ptr, row_ptr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ind, col_ind, num_nonzeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, num_nonzeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, num_cols * sizeof(int), hipMemcpyHostToDevice);

    int block_size = 256;
    int num_blocks = (num_rows + block_size - 1) / block_size;

    spmv_kernel<<<num_blocks, block_size>>>(d_row_ptr, d_col_ind, d_values, d_x, d_y, num_rows);

    hipMemcpy(y, d_y, num_rows * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i = 0; i < num_rows; i++) 
        printf("%d\n", y[i]);

    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
